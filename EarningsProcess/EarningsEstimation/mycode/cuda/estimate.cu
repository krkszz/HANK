#include "hip/hip_runtime.h"
#include "helper_timer.h" // helper functions for timers
#include "hip/hip_runtime_api.h"      // helper functions (cuda error checking and initialization)
#include "newuoa_h.h"
#include <hiprand/hiprand_kernel.h>
// Thread block size
#define THREAD_N 128

__device__ double4 reduce_sum(double4 in, int n) {
	extern __shared__ double4 sdata[];

	// Perform first level of reduction:
	// - Write to shared memory
	int ltid = threadIdx.x;

	sdata[ltid] = in;
	__syncthreads();

	// Do reduction in shared mem
	for (int s = blockDim.x / 2 ; s > 0 ; s >>= 1) {
		if (ltid < s) {
			double d = sdata[ltid + s].x - sdata[ltid].x, dn = d / 2, dn2 = dn * dn, d2 = d * dn;
			sdata[ltid].w += sdata[ltid + s].w + d2 * dn2 * n + 6 * dn2 * (sdata[ltid].y + sdata[ltid + s].y) - 4 * dn * (sdata[ltid].z - sdata[ltid + s].z);
			sdata[ltid].z += sdata[ltid + s].z - 3 * dn * (sdata[ltid].y - sdata[ltid + s].y);
			sdata[ltid].y += sdata[ltid + s].y + d2 * n;
			sdata[ltid].x += dn;
			n <<= 1;
		}
		__syncthreads();
	}

	return sdata[0];
}

__device__ double4 reduce_fractions(double4 in) {
	extern __shared__ double4 sdata[];

	// Perform first level of reduction:
	// - Write to shared memory
	int ltid = threadIdx.x;

	sdata[ltid] = in;
	__syncthreads();

	// Do reduction in shared mem
	for (int s = blockDim.x / 2 ; s > 0 ; s >>= 1) {
		if (ltid < s) {
			sdata[ltid].x += (sdata[ltid + s].x - sdata[ltid].x) / 2;
			sdata[ltid].y += (sdata[ltid + s].y - sdata[ltid].y) / 2;
			sdata[ltid].z += (sdata[ltid + s].z - sdata[ltid].z) / 2;
			sdata[ltid].w += (sdata[ltid + s].w - sdata[ltid].w) / 2;
		}
		__syncthreads();
	}

	return sdata[0];
}
__device__ inline void computeMoments(double4 *m, double x, int n) {
	double d, d2, dn, dn2;

	d = x - m->x;
	dn = d / (n + 1);
	dn2 = dn * dn;
	d2 = d * dn * n;
	m->w += d2 * dn2 * (n*n - n + 1) + 6 * dn2 * m->y - 4 * dn * m->z;
	m->z += d2 * dn * (n - 1) - 3 * dn * m->y;
	m->y += d2;
	m->x += dn;
}

__device__ inline void computeFractions(double4 *m, double x, int n) {
	m->x += ((x < 0.05) - m->x) / (n + 1);
	m->y += ((x < 0.1) - m->y) / (n + 1);
	m->z += ((x < 0.2) - m->z) / (n + 1);
	m->w += ((x < 0.5) - m->w) / (n + 1);
}

// Simulation kernel
__launch_bounds__(1024)
__global__ void simulate(hiprandState *const rngStates1, hiprandStatePhilox4_32_10 *const rngStates2,
	double4* moments, const int nsim, const double2 lambda, const double2 sigma, const double2 delta) {

	// Determine thread ID
	int bid = blockIdx.x;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int step = gridDim.x * blockDim.x; 
	double4 m[4] = { make_double4(0,0,0,0) };

	// Initialise the RNG
	hiprandState state1 = rngStates1[tid];
	hiprandStatePhilox4_32_10 state2 = rngStates2[tid];

	for (int i = tid; i < nsim; i += step) {
		// draw initial from normal distribution with same mean and variance
		double2 z = hiprand_normal2_double(&state1);
		z.x = sigma.x/sqrt(1+2*delta.x/lambda.x)*z.x;
		z.y = sigma.y/sqrt(1+2*delta.y/lambda.y)*z.y;

		// simulate income path in dt increments
		double zann[5] = { 0.0 };
		for (int t=-25; t<5; t++) // burn 25 years, only need 5 years
			for (int q=0; q<16; q++) {
				// Generate pseudo-random numbers
				double2 rand = hiprand_normal2_double(&state1);
				double2 jumprand = hiprand_uniform2_double(&state2);
				z.x = jumprand.x > 1 - lambda.x/4 ? sigma.x*rand.x : (1 - delta.x/4) * z.x;
				z.y = jumprand.y > 1 - lambda.y/4 ? sigma.y*rand.y : (1 - delta.y/4) * z.y;
				if (t >= 0) zann[t] += exp(z.x + z.y); // aggregate to annual income
			}

//if (tid == 0) printf("%d/%d% d/%d: %.15g %.15g %.15g\n",threadIdx.x,blockDim.x,blockIdx.x,gridDim.x,log(zann[0]),log(zann[1]/zann[0]),log(zann[4]/zann[0]));
		// Compute central moments
		computeMoments(&m[0],log(zann[0]),i/step); // logs
		computeMoments(&m[1],log(zann[1]/zann[0]),i/step); // 1 year log changes
		computeMoments(&m[2],log(zann[4]/zann[0]),i/step); // 5 year log changes
		computeFractions(&m[3],abs(log(zann[1]/zann[0])),i/step); // fraction 1 year log changes in ranges
	}
//if (blockIdx.x==0) printf("%03d: %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g\n",tid,m[0].x,m[0].y,m[0].z,m[0].w,m[1].x,m[1].y,m[1].z,m[1].w,m[2].x,m[2].y,m[2].z,m[2].w);

	// Copy RNG state back to global memory
//	rngStates1[tid] = state1;
//	rngStates2[tid] = state2;

	// Reduce within the block
	m[0] = reduce_sum(m[0],nsim/step);
	m[1] = reduce_sum(m[1],nsim/step);
	m[2] = reduce_sum(m[2],nsim/step);
	m[3] = reduce_fractions(m[3]);

	// Store the result
	if (threadIdx.x == 0) {
		moments[bid*4] = m[0];
		moments[bid*4+1] = m[1];
		moments[bid*4+2] = m[2];
		moments[bid*4+3] = m[3];
//printf("%03d: %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g %.15g\n",tid,m[0].x,m[0].y,m[0].z,m[0].w,m[1].x,m[1].y,m[1].z,m[1].w,m[2].x,m[2].y,m[2].z,m[2].w);
	}
}

// RNG init kernel
static __global__ void rngSetupStates(hiprandState *const rngStates1, hiprandStatePhilox4_32_10 *const rngStates2, int device_id) {
	// Determine global thread ID
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// Each threadblock get different seed,
	// Threads within a threadblock get different sequence numbers
	const int seed = blockIdx.x + gridDim.x * device_id; //(int)clock64();
	// Initialise the RNG
	hiprand_init(seed, tid, 0, &rngStates1[tid]);
	hiprand_init(seed, tid, 0, &rngStates2[tid]);
}

typedef struct PlanType
{
	// Device ID for multi-GPU version
	int device;
	// Simulation path count for this plan
	int nsim;
	int gridSize;
	// Stream handle and event object for this plan
	hipStream_t stream;
	hipEvent_t event;
	// Device- and host-side intermediate results
	double4 *d_moments;
	double4 *h_moments;
	// Random number generator states
	hiprandState *d_rngStates1;
	hiprandStatePhilox4_32_10 *d_rngStates2;
} PlanType;

typedef struct UserParamsType {
	int nf;
	int nPlans;
	PlanType *plan;
	// Host-side target moments and result destination
	double4 targets[4];
	double4 moments[4];
} UserParamsType;

static void dfovec(const long int nx, const long int mv, const double *x, double *v_err, const void * userParams) {
	UserParamsType *pUserParams = (UserParamsType *) userParams;
	PlanType *plan = pUserParams->plan;
	int nPlans = pUserParams->nPlans;
	double4 *targets = pUserParams->targets;
	double4 *moments = pUserParams->moments;
	double2 lambda = make_double2(2 / (1 + exp(-x[0])), 2 / (1 + exp(-x[1])));
	double2 sigma = make_double2(2 / (1 + exp(-x[2])), 2 / (1 + exp(-x[3])));
	double2 delta = make_double2(1 / (1 + exp(-x[4])), 1 / (1 + exp(-x[5])));

	if (nx != 6 || mv != 8) {
		fprintf(stderr,"*** dfovec incorrectly called with n=%d and mv=%d\n",nx,mv);
		return;
	}

	for (int i=0; i<nPlans; i++) {
		// Simulate the process and compute moments
		checkCudaErrors(hipSetDevice(plan[i].device));
		simulate<<<plan[i].gridSize, THREAD_N, THREAD_N*sizeof(double4), plan[i].stream>>>(plan[i].d_rngStates1,  plan[i].d_rngStates2, plan[i].d_moments, plan[i].nsim, lambda, sigma, delta);
		getLastCudaError("Failed to launch simulate kernel\n");

		// Copy partial results to host
		checkCudaErrors(hipMemcpyAsync(plan[i].h_moments, plan[i].d_moments, plan[i].gridSize*4*sizeof(double4), hipMemcpyDeviceToHost, plan[i].stream));

		checkCudaErrors(hipEventRecord(plan[i].event, plan[i].stream));
	}
	for (int i=0; i<nPlans; i++) {
		checkCudaErrors(hipSetDevice(plan[i].device));
		hipEventSynchronize(plan[i].event);
	}

	// Complete reduction on host
	for (int j=0; j<3; j++) {
		double	m1 = 0, m2 = 0, m3 = 0, m4 = 0;
		int nsim = 0;
		for (int i=0; i<nPlans; i++) {
			int nb = plan[i].nsim / plan[i].gridSize;
			for (int n=0; n<plan[i].gridSize; n++) {
				double4 m = plan[i].h_moments[n*4+j];
				double d = m.x - m1, dn = d / (nsim + nb), dn2 = dn * dn, d2 = d * dn * nb * nsim;
				m4 += m.w + d2 * dn2 * (nsim*nsim - nsim*nb + nb*nb) + 6 * dn2 * (nsim*nsim*m.y + nb*nb*m2) + 4 * dn * (nsim*m.z - nb*m3);
				m3 += m.z + d2 * dn * (nsim - nb) + 3 * dn * (nsim*m.y - nb*m2);
				m2 += m.y + d2;
				m1 += dn * nb;
				nsim += nb;
//printf("++ %.15g %.15g %.15g %.15g\n",m.x,m.y,m.z,m.w);
			}
//printf("%.15g %.15g %.15g %.15g\n",m1,m2,m3,m4);
		}
		// Compute standardised moments
		m2 /= nsim;
		m3 /= nsim*m2*sqrt(m2);
		m4 /= nsim*m2*m2;
		moments[j].x = m1; //mean
		moments[j].y = m2; // variance
		moments[j].z = m3; // skewness
		moments[j].w = m4; // kurtosis
//printf("%.15g %.15g %.15g %.15g\n",moments[j].x,moments[j].y,moments[j].z,moments[j].w);
	}
	// Compute fraction of dy1 less than 5%, 10%, 20% and 50%
	moments[3] = make_double4(0.0,0.0,0.0,0.0);
	int nsim = 0;
	for (int i=0; i<nPlans; i++) {
		int nb = plan[i].nsim / plan[i].gridSize;
		for (int n=0; n<plan[i].gridSize; n++) {
			double4 m = plan[i].h_moments[n*4+3];
			moments[3].x += (m.x - moments[3].x) * nb / (nsim + nb);
			moments[3].y += (m.y - moments[3].y) * nb / (nsim + nb);
			moments[3].z += (m.z - moments[3].z) * nb / (nsim + nb);
			moments[3].w += (m.w - moments[3].w) * nb / (nsim + nb);
			nsim += nb;
		}
	}
//printf("%.15g %.15g %.15g %.15g\n",moments[3].x,moments[3].y,moments[3].z,moments[3].w);

//	printf("%.15g\t%.15g\t%.15g\t%.15g\t%.15g\t%.15g\t%.15g\n",obj,lambda.x,lambda.y,sigma.x,sigma.y,delta.x,delta.y);
	v_err[0] = moments[0].y/targets[0].y-1;
	v_err[1] = moments[1].y/targets[1].y-1;
	v_err[2] = moments[1].w/targets[1].w-1;
	v_err[3] = moments[2].y/targets[2].y-1;
	v_err[4] = moments[2].w/targets[2].w-1;
	v_err[5] = moments[3].y/targets[3].y-1;
	v_err[6] = moments[3].z/targets[3].z-1;
	v_err[7] = moments[3].w/targets[3].w-1;
	v_err[2] *= sqrt(0.5);
	v_err[4] *= sqrt(0.5);

	++pUserParams->nf;
}

int main(int argc, char *argv[]) {
	// Get number of available devices
	int GPU_N = 0;
	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	if (!GPU_N) {
		fprintf(stderr,"There are no CUDA devices.\n");
		exit(EXIT_FAILURE);
	}
	printf("CUDA-capable device count: %i\n", GPU_N);

	long NSIM = 1;
	if (argc<=1) {
		fprintf(stderr,"Usage: estimate N, where N is the exponent of two in the number of simulation paths.\n");
		exit(EXIT_FAILURE);
	} else
		NSIM <<= atoi(argv[1]);
	if (((NSIM/GPU_N) % THREAD_N) | (NSIM < GPU_N)) {
		fprintf(stderr,"The number of simulation paths per GPU must be a multiple of block size %d.\n",THREAD_N);
		exit(EXIT_FAILURE);
	}

	UserParamsType userParams;
	userParams.nf = 0;
	userParams.nPlans = GPU_N;
	userParams.plan = new PlanType[GPU_N];
	for (int device=0; device<GPU_N; device++) {
		// Attach to GPU
		checkCudaErrors(hipSetDevice(device));
		// Get device properties
		struct hipDeviceProp_t deviceProperties;
		checkCudaErrors(hipGetDeviceProperties(&deviceProperties, device));
		// Check precision is valid
		if (deviceProperties.major < 1 || (deviceProperties.major == 1 && deviceProperties.minor < 3)) {
			printf("Device %d does not have double precision support.\n", device);
			exit(EXIT_FAILURE);
		}

		PlanType *p = &userParams.plan[device];
		p->device = device;

		// Initialize stream handle and event object for the current device
		checkCudaErrors(hipStreamCreate(&p->stream));
		checkCudaErrors(hipEventCreate(&p->event));

		// Divide the work between GPUs equally
		p->nsim = NSIM / GPU_N;
		if (device < (NSIM % GPU_N)) p->nsim++;

		p->gridSize = p->nsim / THREAD_N;
		// Aim to launch around ten to twenty times as many blocks as there
		// are multiprocessors on the target device.
		// read more on grid-stride loops: https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
		while (p->gridSize > 20 * deviceProperties.multiProcessorCount) p->gridSize >>= 1;

		printf("GPU Device #%i: %s\n", p->device, deviceProperties.name);
		printf("Simulation paths: %i\n", p->nsim);
		printf("Grid size: %i\n", p->gridSize);

		// Allocate intermediate memory for MC results
		// Each thread block will produce four double4 results
		checkCudaErrors(hipHostMalloc(&p->h_moments,p->gridSize*4*sizeof(double4)));
		checkCudaErrors(hipMalloc(&p->d_moments, p->gridSize*4*sizeof(double4)));

		// Allocate memory for RNG states
		checkCudaErrors(hipMalloc(&p->d_rngStates1, p->gridSize * THREAD_N * sizeof(hiprandState)));
		checkCudaErrors(hipMalloc(&p->d_rngStates2, p->gridSize * THREAD_N * sizeof(hiprandStatePhilox4_32_10)));
		// Initialise RNG states so that each device is placed pathN random numbers apart on the random number sequence
		rngSetupStates<<<p->gridSize, THREAD_N>>>(p->d_rngStates1, p->d_rngStates2, p->device);
		getLastCudaError("rngSetupStates kernel failed.\n");
		checkCudaErrors(hipDeviceSynchronize());
	}

	// Target moments for USA: 0.7,0.23,17.8,0.46,11.55,0.54,0.71,0.86
	// Target moments for Canada: 0.760,0.217,13.377,0.437,8.782,0.51,0.68,0.85
	userParams.targets[0] = make_double4(NAN, 0.760, NAN, NAN); // LogY: Mean,Var,Skew,Kurt
	userParams.targets[1] = make_double4(NAN, 0.217, NAN, 13.377); // D1LogY: Mean,Var,Skew,Kurt
	userParams.targets[2] = make_double4(NAN, 0.437, NAN, 8.782); // D5LogY: Mean,Var,Skew,Kurt
	userParams.targets[3] = make_double4(NAN, 0.51, 0.68, 0.85); // FracD1: <5%,<10%,<20%,<50%

	long int n=6, mv=8, npt=2*n+1, maxfun=500*(n+1), iprint=1;
	double v_err[8], rhobeg=5.0, rhoend=1e-4, *w;
	double xmax[6] = {2,2,2,2,1,1}, xmin[6] = {0};
//	double x[6] = {0.0972241396763905,  0.014312611368279, 1.60304896242711, 0.892309166034993, 0.947420941274568,  0.00117609031021279};
	double x[6] = {.08,.007,1.6,1.6,.7,.01};
//	double x[6] = {0.0611244618471226,0.000613274511999765,1.46320215181056,1.999691573564,0.224227629475885,0.0018853181294203};

	int wsize = (npt+11)*(npt+n)+n*(5*n+11)/2+mv*(npt+n*(n+7)/2+7);
	checkCudaErrors(hipHostMalloc(&w,wsize*sizeof(double)));

	for (int i = 0; i<6; i++)
		x[i] = -log(xmax[i] / (x[i] - xmin[i]) - 1); // invlogistic

	//Start the timer
	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);
	sdkResetTimer(&hTimer);
	sdkStartTimer(&hTimer);

	newuoa_h(n, npt, dfovec, &userParams, x, rhobeg, rhoend, iprint, maxfun, w, mv);
	dfovec(n,mv,x,v_err,&userParams);

	//Stop the timer
	sdkStopTimer(&hTimer);
	float time = sdkGetTimerValue(&hTimer)/userParams.nf;
	sdkDeleteTimer(&hTimer);

	double obj = 0;
	for (int i=0; i<mv; i++)
		obj += v_err[i]*v_err[i];

	for (int i=0; i<6; i++)
		x[i] = xmin[i]+xmax[i]/(1+exp(-x[i])); // logistic

	printf("\nTime per function evaluation (ms.): %f\n", time);
	printf("\nFinal objective function value: %.15g\n",obj);//sqrt(obj*2/7));
	printf("\nThe returned solution is:\n");
	printf(" lambda: %.15g  %.15g\n",x[0],x[1]);
	printf(" sigma:  %.15g  %.15g\n",x[2],x[3]);
	printf(" delta:  %.15g  %.15g\n",x[4],x[5]);
	printf("\n Moment:      Target:\tModel:\n");
	printf(" MeanLogY     %.15g\t%.15g\n",userParams.targets[0].x,userParams.moments[0].x);
	printf(" VarLogY      %.15g\t%.15g\n",userParams.targets[0].y,userParams.moments[0].y);
	printf(" SkewLogY     %.15g\t%.15g\n",userParams.targets[0].z,userParams.moments[0].z);
	printf(" KurtLogY     %.15g\t%.15g\n",userParams.targets[0].w,userParams.moments[0].w);
	printf(" MeanD1LogY   %.15g\t%.15g\n",userParams.targets[1].x,userParams.moments[1].x);
	printf(" VarD1LogY    %.15g\t%.15g\n",userParams.targets[1].y,userParams.moments[1].y);
	printf(" SkewD1LogY   %.15g\t%.15g\n",userParams.targets[1].z,userParams.moments[1].z);
	printf(" KurtD1LogY   %.15g\t%.15g\n",userParams.targets[1].w,userParams.moments[1].w);
	printf(" MeanD5LogY   %.15g\t%.15g\n",userParams.targets[2].x,userParams.moments[2].x);
	printf(" VarD5LogY    %.15g\t%.15g\n",userParams.targets[2].y,userParams.moments[2].y);
	printf(" SkewD5LogY   %.15g\t%.15g\n",userParams.targets[2].z,userParams.moments[2].z);
	printf(" KurtD5LogY   %.15g\t%.15g\n",userParams.targets[2].w,userParams.moments[2].w);
	printf(" FracD1Less5  %.15g\t%.15g\n",userParams.targets[3].x,userParams.moments[3].x);
	printf(" FracD1Less10 %.15g\t%.15g\n",userParams.targets[3].y,userParams.moments[3].y);
	printf(" FracD1Less20 %.15g\t%.15g\n",userParams.targets[3].z,userParams.moments[3].z);
	printf(" FracD1Less50 %.15g\t%.15g\n",userParams.targets[3].w,userParams.moments[3].w);

	// Cleanup
	for (int device=0; device<GPU_N; device++) {
		PlanType *p = &userParams.plan[device];
		checkCudaErrors(hipSetDevice(p->device));
		checkCudaErrors(hipStreamDestroy(p->stream));
		checkCudaErrors(hipEventDestroy(p->event));
		checkCudaErrors(hipHostFree(p->h_moments));
		checkCudaErrors(hipFree(p->d_moments));
		checkCudaErrors(hipFree(p->d_rngStates1));
		checkCudaErrors(hipFree(p->d_rngStates2));
	}
	checkCudaErrors(hipHostFree(w));
	delete[] userParams.plan;
	return(0);
}

